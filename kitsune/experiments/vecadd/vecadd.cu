#include "hip/hip_runtime.h"
#include "kitsune/timer.h"

#include <hip/hip_runtime.h>
#include <float.h>
#include <fstream>
#include <iostream>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <time.h>

const size_t VEC_SIZE = 1024 * 1024 * 256;

enum PrefetchKinds {
  EXPLICIT = 0,     // Use explicit async prefetch calls. 
  PRELAUNCH = 1,    // Prelaunch the kernel to move pages to device. 
  NONE = 2          // Do nothing, default to built-in page management. 
};

void random_fill(float *data, size_t N) {
  for (size_t i = 0; i < N; ++i)
    data[i] = rand() / (float)RAND_MAX;
}


__global__ void VectorAdd(float *A, float *B, float *C, size_t N) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}


int main(int argc, char *argv[]) {
  size_t size = VEC_SIZE;
  PrefetchKinds PFKind = NONE;

  if (argc > 1 ) {
    size = atol(argv[1]);
    if (argc == 3) {
      if (std::string(argv[2]) == "explicit")
        PFKind = EXPLICIT;
      else if (std::string(argv[2]) == "pre-launch")
        PFKind = PRELAUNCH;
      else
        PFKind = NONE;
    }
  }

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  hipEvent_t start;
  hipEventCreate(&start);
  hipEventRecord(start);
  hipError_t err = hipSuccess;
  float *A, *B, *C;
  err = hipMallocManaged(&A, size * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory for A!\n");
    return 1;
  }
  err = hipMallocManaged(&B, size * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory for B!\n");
    return 1;
  }
  err = hipMallocManaged(&C, size * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory for C!\n");
    return 1;
  }

  if (PFKind == EXPLICIT)
    hipMemPrefetchAsync(C, sizeof(float) * size, 0, nullptr);

  random_fill(A, size);
  if (PFKind == EXPLICIT)
    hipMemPrefetchAsync(A, sizeof(float) * size, 0, nullptr);

  random_fill(B, size);
  if (PFKind == EXPLICIT)
    hipMemPrefetchAsync(B, sizeof(float) * size, 0, nullptr);

  if (PFKind == PRELAUNCH) {
    // prime the GPU...  This will move all data to the device
    // prior to the timed launch below....
    VectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, size);
    //hipDeviceSynchronize();
  }

  hipEvent_t kstart, kstop;
  hipEventCreate(&kstart);
  hipEventCreate(&kstop);


  hipEventRecord(kstart);
  VectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, size);
  hipEventRecord(kstop);
  hipEventSynchronize(kstop);

  float msecs = 0;
  hipEventElapsedTime(&msecs, kstart, kstop);
  printf("%.8g\n", msecs / 1000.0);

  hipEventElapsedTime(&msecs, start, kstop);
  fprintf(stderr, "%.8lg\n", msecs / 1000.0);

  // Sanity check the results...
  size_t error_count = 0;
  for (size_t i = 0; i < size; i++) {
    float sum = A[i] + B[i];
    if (C[i] != sum)
      error_count++;
  }

  if (error_count != 0)
    printf("bad result!\n");

  return 0;
}
